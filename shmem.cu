// Shared memory bandwidth benchmark
// contributed by Sebastian Keller
// 
// Reproduces the theoretical NVidia figures quite exactly
// tested on P100 and V100
//
// Relevant nvprof metrics:
// nvprof -m shared_load_throughput,shared_store_throughput

#include <cstdint>
#include <iostream>

#include <malloc.h>
#include <hip/hip_runtime.h>



#define NTHREADS 256
#define NITER    4096
// length of the thread block swap chain (must be even)
#define SHARED_SEGMENTS 4

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

template <class T>
__device__ void swap(T* a, T* b)
{
    T tmp;
    tmp = *a;
    *a = *b;
    *b = tmp;
}

template <class T>
__global__ void test_shmem(T* glob_mem)
{
    volatile __shared__ T smem[NTHREADS*SHARED_SEGMENTS];

    int tid = threadIdx.x;

    smem[tid] = T{0};
    for (int i = 0; i < NITER; ++i)
    {
        // even shared segments
        for (int j = 0; j < SHARED_SEGMENTS-1; j+=2)
            swap(smem + tid + j*blockDim.x, smem + tid + (j+1)*blockDim.x);

        // uneven shared segments
        for (int j = 1; j < SHARED_SEGMENTS-2; j+=2)
            swap(smem + tid + j*blockDim.x, smem + tid + (j+1)*blockDim.x);
    }

    glob_mem[blockIdx.x * blockDim.x + tid] = smem[tid];
}

template <class T>
double test_bw(long size)
{
    T* buffer = (T*)malloc(size);
    T* dev_buffer;
    HANDLE_ERROR( hipMalloc((void**)&dev_buffer, size) );
    int nblocks = size / (NTHREADS * sizeof(T));

    // call once
    test_shmem<<<nblocks, NTHREADS>>>(dev_buffer);

    hipEvent_t start, stop;
    HANDLE_ERROR( hipEventCreate(&start) );
    HANDLE_ERROR( hipEventCreate(&stop) );
    HANDLE_ERROR( hipEventRecord(start,0) );

    test_shmem<<<nblocks, NTHREADS>>>(dev_buffer);

    HANDLE_ERROR( hipEventRecord(stop,0) );
    HANDLE_ERROR( hipEventSynchronize(stop) );
    float gpu_time;
    HANDLE_ERROR( hipEventElapsedTime( &gpu_time, start, stop ) );
    // convert to seconds
    gpu_time /= 1000;

    // 2 writes + 2 reads per swap
    double nbytes = NITER * size * (SHARED_SEGMENTS-1) * 4;

    hipEventDestroy(start);
    hipEventDestroy(stop);
    free(buffer);
    hipFree(dev_buffer);

    return nbytes / gpu_time;
}

int main()
{
    long size = 1024 * 1024 * 64; // 64 MB global buffer

    // warmup
    test_bw<int>(size);


    std::cout << "Bandwidth(int) " << test_bw<int>(size) / 1024 / 1024 / 1024 << " GB/s" << std::endl;
    std::cout << "Bandwidth(int64) " << test_bw<int64_t>(size) / 1024 / 1024 / 1024 << " GB/s" << std::endl;
}
  
